#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
// #include <hip/device_functions.h>

#define CHECK_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
    if(code != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if(abort) exit(code);
    }
}

__device__ inline uint32_t F(uint32_t x, uint32_t y, uint32_t z) {
    return z ^ (x & (y ^ z));
}
 
__device__ inline uint32_t G(uint32_t x, uint32_t y, uint32_t z) {
    return y ^ (z & (x ^ y));
}
 
__device__ inline uint32_t H(uint32_t x, uint32_t y, uint32_t z) {
    return x ^ y ^ z;
}
 
__device__ inline uint32_t I(uint32_t x, uint32_t y, uint32_t z) {
    return y ^ (x | ~z);
}
 
__device__ inline uint32_t rotate_left(uint32_t x, int n) {
    return (x << n) | (x >> (32-n));
}

__device__ inline void FF(uint32_t& a, uint32_t b, uint32_t c, uint32_t d, uint32_t x, uint32_t s, uint32_t ac) {
    a = rotate_left(a + F(b,c,d) + x + ac, s) + b;
}

__device__ inline void GG(uint32_t& a, uint32_t b, uint32_t c, uint32_t d, uint32_t x, uint32_t s, uint32_t ac) {
    a = rotate_left(a + G(b,c,d) + x + ac, s) + b;
}

__device__ inline void HH(uint32_t& a, uint32_t b, uint32_t c, uint32_t d, uint32_t x, uint32_t s, uint32_t ac) {
    a = rotate_left(a + H(b,c,d) + x + ac, s) + b;
}

__device__ inline void II(uint32_t& a, uint32_t b, uint32_t c, uint32_t d, uint32_t x, uint32_t s, uint32_t ac) {
    a = rotate_left(a + I(b,c,d) + x + ac, s) + b;
}

__device__ inline void padding(uint32_t* x, unsigned char data[], uint32_t length) {
    // padding the input string
    int i = 0;
    for(i=0; i < length; i++){
        x[i / 4] |= data[i] << ((i % 4) * 8);
    }
    
    x[i / 4] |= 0x80 << ((i % 4) * 8);

    uint32_t bitlen = length * 8;
    x[14] = bitlen;
    x[15] = 0;
}

__device__ inline void md5Hash(unsigned char* data, uint32_t length, uint32_t *a1, uint32_t *b1, uint32_t *c1, uint32_t *d1){
    const uint32_t a0 = 0x67452301;
    const uint32_t b0 = 0xEFCDAB89;
    const uint32_t c0 = 0x98BADCFE;
    const uint32_t d0 = 0x10325476;

    uint32_t a = 0;
    uint32_t b = 0;
    uint32_t c = 0;
    uint32_t d = 0;
    
    // padding the input string
    uint32_t x[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    padding(x, data, length);
    // int i = 0;
    // for(i=0; i < length; i++){ 
    //     x[i / 4] |= data[i] << ((i % 4) * 8);
    // }
    
    // x[i / 4] |= 0x80 << ((i % 4) * 8);

    // uint32_t bitlen = length * 8;
    // x[14] = bitlen;
    // x[15] = 0;

    //Initialize hash value for this chunk:
    a = a0;
    b = b0;
    c = c0;
    d = d0;

    /* Round 1 */
    #define S11 7
    #define S12 12
    #define S13 17
    #define S14 22
    FF (a, b, c, d, x[ 0], S11, 0xd76aa478); // 1
    FF (d, a, b, c, x[ 1], S12, 0xe8c7b756); // 2
    FF (c, d, a, b, x[ 2], S13, 0x242070db); // 3
    FF (b, c, d, a, x[ 3], S14, 0xc1bdceee); // 4
    FF (a, b, c, d, x[ 4], S11, 0xf57c0faf); // 5
    FF (d, a, b, c, x[ 5], S12, 0x4787c62a); // 6
    FF (c, d, a, b, x[ 6], S13, 0xa8304613); // 7
    FF (b, c, d, a, x[ 7], S14, 0xfd469501); // 8
    FF (a, b, c, d, x[ 8], S11, 0x698098d8); // 9
    FF (d, a, b, c, x[ 9], S12, 0x8b44f7af); // 10
    FF (c, d, a, b, x[10], S13, 0xffff5bb1); // 11
    FF (b, c, d, a, x[11], S14, 0x895cd7be); // 12
    FF (a, b, c, d, x[12], S11, 0x6b901122); // 13
    FF (d, a, b, c, x[13], S12, 0xfd987193); // 14
    FF (c, d, a, b, x[14], S13, 0xa679438e); // 15
    FF (b, c, d, a, x[15], S14, 0x49b40821); // 16

    /* Round 2 */
    #define S21 5
    #define S22 9
    #define S23 14
    #define S24 20
    GG (a, b, c, d, x[ 1], S21, 0xf61e2562); // 17
    GG (d, a, b, c, x[ 6], S22, 0xc040b340); // 18
    GG (c, d, a, b, x[11], S23, 0x265e5a51); // 19
    GG (b, c, d, a, x[ 0], S24, 0xe9b6c7aa); // 20
    GG (a, b, c, d, x[ 5], S21, 0xd62f105d); // 21
    GG (d, a, b, c, x[10], S22,  0x2441453); // 22
    GG (c, d, a, b, x[15], S23, 0xd8a1e681); // 23
    GG (b, c, d, a, x[ 4], S24, 0xe7d3fbc8); // 24
    GG (a, b, c, d, x[ 9], S21, 0x21e1cde6); // 25
    GG (d, a, b, c, x[14], S22, 0xc33707d6); // 26
    GG (c, d, a, b, x[ 3], S23, 0xf4d50d87); // 27
    GG (b, c, d, a, x[ 8], S24, 0x455a14ed); // 28
    GG (a, b, c, d, x[13], S21, 0xa9e3e905); // 29
    GG (d, a, b, c, x[ 2], S22, 0xfcefa3f8); // 30
    GG (c, d, a, b, x[ 7], S23, 0x676f02d9); // 31
    GG (b, c, d, a, x[12], S24, 0x8d2a4c8a); // 32

    /* Round 3 */
    #define S31 4
    #define S32 11
    #define S33 16
    #define S34 23
    HH (a, b, c, d, x[ 5], S31, 0xfffa3942); // 33
    HH (d, a, b, c, x[ 8], S32, 0x8771f681); // 34
    HH (c, d, a, b, x[11], S33, 0x6d9d6122); // 35
    HH (b, c, d, a, x[14], S34, 0xfde5380c); // 36
    HH (a, b, c, d, x[ 1], S31, 0xa4beea44); // 37
    HH (d, a, b, c, x[ 4], S32, 0x4bdecfa9); // 38
    HH (c, d, a, b, x[ 7], S33, 0xf6bb4b60); // 39
    HH (b, c, d, a, x[10], S34, 0xbebfbc70); // 40
    HH (a, b, c, d, x[13], S31, 0x289b7ec6); // 41
    HH (d, a, b, c, x[ 0], S32, 0xeaa127fa); // 42
    HH (c, d, a, b, x[ 3], S33, 0xd4ef3085); // 43
    HH (b, c, d, a, x[ 6], S34,  0x4881d05); // 44
    HH (a, b, c, d, x[ 9], S31, 0xd9d4d039); // 45
    HH (d, a, b, c, x[12], S32, 0xe6db99e5); // 46
    HH (c, d, a, b, x[15], S33, 0x1fa27cf8); // 47
    HH (b, c, d, a, x[ 2], S34, 0xc4ac5665); // 48
    
    /* Round 4 */
    #define S41 6
    #define S42 10
    #define S43 15
    #define S44 21
    II (a, b, c, d, x[ 0], S41, 0xf4292244); // 49
    II (d, a, b, c, x[ 7], S42, 0x432aff97); // 50
    II (c, d, a, b, x[14], S43, 0xab9423a7); // 51
    II (b, c, d, a, x[ 5], S44, 0xfc93a039); // 52
    II (a, b, c, d, x[12], S41, 0x655b59c3); // 53
    II (d, a, b, c, x[ 3], S42, 0x8f0ccc92); // 54
    II (c, d, a, b, x[10], S43, 0xffeff47d); // 55
    II (b, c, d, a, x[ 1], S44, 0x85845dd1); // 56
    II (a, b, c, d, x[ 8], S41, 0x6fa87e4f); // 57
    II (d, a, b, c, x[15], S42, 0xfe2ce6e0); // 58
    II (c, d, a, b, x[ 6], S43, 0xa3014314); // 59
    II (b, c, d, a, x[13], S44, 0x4e0811a1); // 60
    II (a, b, c, d, x[ 4], S41, 0xf7537e82); // 61
    II (d, a, b, c, x[11], S42, 0xbd3af235); // 62
    II (c, d, a, b, x[ 2], S43, 0x2ad7d2bb); // 63
    II (b, c, d, a, x[ 9], S44, 0xeb86d391); // 64

    a += a0;
    b += b0;
    c += c0;
    d += d0;

    *a1 = a;
    *b1 = b;
    *c1 = c;
    *d1 = d;
}